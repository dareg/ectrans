#define cufftSafeCall(err) __cufftSafeCall(err, __FILE__, __LINE__)

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

static const char *_cudaGetErrorEnum(hipfftResult error)
{
  switch (error)
    {
#define cr(x) case CUFFT_##x: return #x
      cr (SUCCESS);
      cr (INVALID_PLAN);
      cr (ALLOC_FAILED);
      cr (INVALID_TYPE);
      cr (INVALID_VALUE);
      cr (INTERNAL_ERROR);
      cr (EXEC_FAILED);
      cr (SETUP_FAILED);
      cr (INVALID_SIZE);
      cr (UNALIGNED_DATA);
#undef cr
    }
  return "UNKNOWN";
}

inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
{
  if (HIPFFT_SUCCESS != err) 
    {
      fprintf(stderr, "CUFFT at '%s:%d'\n", file, line);
      fprintf(stderr, "CUFFT error %d: %s\nterminating!\n", err, _cudaGetErrorEnum (err));
	  fflush(stderr);
      hipDeviceReset(); 
    } /*else {
		fprintf(stderr, "CUFFT call at %s, %i returned code %s\n",file,line,_cudaGetErrorEnum(err));
		fflush(stderr);
	}*/

}


static int allocatedWorkspace=0;
static void* planWorkspace;
static int planWorkspaceSize=100*1024*1024; //100MB
 
extern "C"
void
//create_plan_fftc_(cufftHandle *PLANp, int *ISIGNp, int *Np, int *LOTp, int *STRIDEp, int *DISTp)
create_plan_fftc_(hipfftHandle *PLANp, int *ISIGNp, int *Np, int *LOTp, int *ISTRIDEp, int *IDISTp, int *OSTRIDEp, int *ODISTp)
{

  int ISIGN = *ISIGNp;
  int N = *Np;
  int LOT = *LOTp;
  
  hipfftHandle plan;
  
  if (hipDeviceSynchronize() != hipSuccess)
    {
      fprintf(stderr, "%s, %i :Cuda error: Failed to synchronize\n",__FILE__,__LINE__);
      return;	
    }
  
  
  // //create a single re-usable workspace
  // if(!allocatedWorkspace){
  //   allocatedWorkspace=1;
  //   //allocate plan workspace
  //   cudaMalloc(&planWorkspace,planWorkspaceSize);
  // }
  //
  // //disable auto allocation so we can re-use a single workspace (created above)
  //  cufftSetAutoAllocation(plan, false);
  
  int embed[1];
  int istride, ostride;
  int idist, odist;
  
  #ifdef TRANS_SINGLE
  cufftType cufft_1 = CUFFT_R2C;
  cufftType cufft_2 = CUFFT_C2R;
  #else
  hipfftType cufft_1 = HIPFFT_D2Z;
  hipfftType cufft_2 = HIPFFT_Z2D;
  #endif
  
  embed[0] = 1;
  istride   = *ISTRIDEp;
  idist     = *IDISTp;
  ostride   = *OSTRIDEp;
  odist     = *ODISTp;
  
  
  cufftSafeCall (hipfftCreate (&plan));
  
/*
  if(0){
    fprintf(stderr,"CreatePlan cuFFT\n","N=",N);
    fprintf(stderr,"%s %d \n","plan=",plan);
    fprintf(stderr,"%s %d \n","LOT=",LOT);
    fprintf(stderr,"%s %d \n","ISIGN=",ISIGN);
    fprintf(stderr,"%s %d \n","Np=",*Np);
    fprintf(stderr,"%s %d \n","ISTRIDEp=",*ISTRIDEp);
    fprintf(stderr,"%s %d \n","IDISTp=",*IDISTp);
    fprintf(stderr,"%s %d \n","OSTRIDEp=",*OSTRIDEp);
    fprintf(stderr,"%s %d \n","ODISTp=",*ODISTp);
    fflush (stderr);
  }
*/ 
  
  hipfftType type = ISIGN < 0 ? cufft_1 : cufft_2;
  
  cufftSafeCall (hipfftPlanMany (&plan, 1, &N, embed, istride, idist, embed, ostride, odist, type, LOT));
  
  if (hipDeviceSynchronize() != hipSuccess)
    {
      fprintf(stderr, "%s, %i :Cuda error: Failed to synchronize\n",__FILE__,__LINE__);
	  fflush(stderr);
      return;	
    }
  
  *PLANp=plan;
  
/*
  fprintf(stderr,"cuFFT plan %i created succesfully\n",plan);
  fflush(stderr);
*/  
  // // get size used by this plan
  // size_t workSize;
  // cufftGetSize(plan,&workSize);
  //
  // // exit if we don't have enough space for the work area in the re-usable workspace
  // if(workSize > planWorkspaceSize){
  //   printf("create_plan_fftc: plan workspace size not large enough - exiting\n");
  // exit(1);
  // }
  
  
  return;

}

